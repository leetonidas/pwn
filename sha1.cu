
#include <hip/hip_runtime.h>
#ifndef uint32_t
#define uint32_t unsigned int
#endif


#define H0 0x67452301
#define H1 0xEFCDAB89
#define H2 0x98BADCFE
#define H3 0x10325476
#define H4 0xC3D2E1F0

__device__
uint32_t rotl(uint32_t x, uint32_t n) {
  return (x >> (32 - n)) | (x << n);
}

__device__
uint32_t get_global_id() {
  uint32_t blockId, threadsPerBlock;
  blockId = blockIdx.z * gridDim.x * gridDim.y
          + blockIdx.y * gridDim.x
          + blockIdx.x;
  threadsPerBlock = blockDim.x;
  return threadIdx.x + threadsPerBlock * blockId;
}

__global__ void crypt_kernel(ulong start, uint32_t *prefix, ulong plen, uint32_t mask, uint32_t *match){
  int t;
  uint32_t W[80], rnd, id, A,B,C,D,E,T1,T2;
  uint32_t Ws[16];

  id = get_global_id();

  //if (id == 0) {
  //  printf("%08x\n", start);
  //}

  // brutforce is build up as: prefix | thr_id:04x | <rnd>:04x | start:08x
  for (t = 0; t < plen; ++t) {
    Ws[t] = prefix[t];
  //  printf("%04x", prefix[t]);
  }
  // printf("%04x\n", id);


  T1 = (id & 0xf) | (((id >> 4) & 0xf) << 8) | (((id >> 8) & 0xf) << 16) | (((id >> 12) & 0xf) << 24);
  T2 = (T1 & 0xe0e0e0e);
  T2 = ((((T2 >> 1) & T2) >> 2) | (((T2 >> 2) & T2) >> 1)) & 0x1010101;
  Ws[plen] = T1 + 0x30303030 + T2 * 0x27;

  T1 = (uint)(start >> 32);
  T1 = (T1 & 0xf) | (((T1 >> 4) & 0xf) << 8) | (((T1 >> 8) & 0xf) << 16) | (((T1 >> 12) & 0xf) << 24);
  T2 = (T1 & 0xe0e0e0e);
  T2 = ((((T2 >> 1) & T2) >> 2) | (((T2 >> 2) & T2) >> 1)) & 0x1010101;
  Ws[plen + 2] = T1 + 0x30303030 + T2 * 0x27;

  T1 = (uint)start;
  T1 = (T1 & 0xf) | (((T1 >> 4) & 0xf) << 8) | (((T1 >> 8) & 0xf) << 16) | (((T1 >> 12) & 0xf) << 24);
  T2 = (T1 & 0xe0e0e0e);
  T2 = ((((T2 >> 1) & T2) >> 2) | (((T2 >> 2) & T2) >> 1)) & 0x1010101;
  Ws[plen + 3] = T1 + 0x30303030 + T2 * 0x27;

  Ws[plen + 4] = 0x80000000;

  for (t = plen + 5; t < 15; ++t) {
    Ws[t] = 0;
  }

  Ws[15] = 128 + 32 * plen;
  // preparing buffer done

  /*
  if (id == 0) {
    printf("%016x: ", start);
    for (t = 0; t < 16; ++t) {
      printf("%08x", Ws[t]);
    }
    printf(" - %u\n", Ws[15]);
  }
  */

  for (rnd = 0; rnd < 0x10000; ++rnd) {
    // uint32_t digest[5];

#pragma unroll
    for (t = 0; t < 16; ++t) {
      W[t] = Ws[t];
    }

    T1 = (rnd & 0xf) | (((rnd >> 4) & 0xf) << 8) | (((rnd >> 8) & 0xf) << 16) | (((rnd >> 12) & 0xf) << 24);
    T2 = (T1 & 0xe0e0e0e);
    T2 = ((((T2 >> 1) & T2) >> 2) | (((T2 >> 2) & T2) >> 1)) & 0x1010101;
    W[plen + 1] = T1 + 0x30303030 + T2 * 0x27;

    for (t = 16; t < 80; t++) {
      W[t] = rotl(W[t-3] ^ W[t-8] ^ W[t-14] ^ W[t-16], 1);
    }

    A = H0;
    B = H1;
    C = H2;
    D = H3;
    E = H4;

    for (t = 0; t < 20; t++) {
      T1 = (B & C) | ((~B) & D);
      T2 = rotl(A, 5) + T1 + E + 0x5A827999 + W[t];
      E = D; D = C; C = rotl(B, 30); B = A; A = T2;
    }

    for (t = 20; t < 40; t++) {
      T1 = B ^ C ^ D;
      T2 = rotl(A, 5) + T1 + E + 0x6ED9EBA1 + W[t];
      E = D; D = C; C = rotl(B, 30); B = A; A = T2;
    }

    for (t = 40; t < 60; t++) {
      T1 = (B & C) | (B & D) | (C & D);
      T2 = rotl(A, 5) + T1 + E + 0x8F1BBCDC + W[t];
      E = D; D = C; C = rotl(B, 30); B = A; A = T2;
    }

    for (t = 60; t < 80; t++) {
      T1 = B ^ C ^ D;
      T2 = rotl(A, 5) + T1 + E + 0xCA62C1D6 + W[t];
      E = D; D = C; C = rotl(B, 30); B = A; A = T2;
    }

    A += H0;

    if ((A & mask) == 0) {
      /*
      for (t = 0; t < 16; ++t) {
        printf("%08x", Ws[t]);
      }
      printf(" - %u\n", Ws[15]);
      */

      match[0] = 1;
      match[1] = id;
      match[2] = rnd;
    }
  }
}